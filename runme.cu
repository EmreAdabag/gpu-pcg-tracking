#include <fstream>
#include <vector>
#include <sstream>
#include <iostream>
#include "toplevel.cuh"
#include "qdldl.h"
#include "rbd_plant.cuh"
#include "settings.cuh"






template <typename T>
std::vector<std::vector<T>> readCSVToVecVec(const std::string& filename) {
    std::vector<std::vector<T>> data;
    std::ifstream infile(filename);

    if (!infile.is_open()) {
        std::cerr << "File [ " << filename << " ] could not be opened!\n";
    } else {
        std::string line;


        while (std::getline(infile, line)) {
            std::vector<T> row;
            std::stringstream ss(line);
            std::string val;

            while (std::getline(ss, val, ',')) {
                row.push_back(std::stof(val));
            }

            data.push_back(row);
        }
    }

    infile.close();
    return data;
}




int main(){

    const uint32_t state_size = grid::NUM_JOINTS*2;
    const uint32_t control_size = grid::NUM_JOINTS;
    const uint32_t knot_points = KNOT_POINTS;
    const pcg_t timestep = .015625;

    const uint32_t traj_test_iters = 1;

    // checks GPU space for pcg
    checkPcgOccupancy<pcg_t>((void *) pcg<pcg_t, state_size, knot_points>, PCG_NUM_THREADS, state_size, knot_points);    
    if(!std::is_same<QDLDL_float, pcg_t>::value){ std::cout << "GPU-PCG QDLDL type mismatch" << std::endl; exit(1); }

    print_test_config();

    const uint32_t recorded_states = 5;
    const uint32_t start_goal_combinations = recorded_states*recorded_states;

    char eePos_traj_file_name[100];
    char xu_traj_file_name[100];

    int start_state, goal_state;
    pcg_t *d_eePos_traj, *d_xu_traj, *d_xs;

    for(uint32_t ind = 0; ind < start_goal_combinations; ind++){

        start_state = ind % recorded_states;
        goal_state = ind / recorded_states;
        if(start_state == goal_state && start_state != 0){ continue; }
        std::cout << "start: " << start_state << " goal: " << goal_state << std::endl;

        for (uint32_t single_traj_test_iter = 0; single_traj_test_iter < traj_test_iters; single_traj_test_iter++){

            // read in traj
            snprintf(eePos_traj_file_name, sizeof(eePos_traj_file_name), "testfiles/%d_%d_eepos.traj", start_state, goal_state);
            std::vector<std::vector<pcg_t>> eePos_traj2d = readCSVToVecVec<pcg_t>(eePos_traj_file_name);
            
            snprintf(xu_traj_file_name, sizeof(xu_traj_file_name), "testfiles/%d_%d_traj.csv", start_state, goal_state);
            std::vector<std::vector<pcg_t>> xu_traj2d = readCSVToVecVec<pcg_t>(xu_traj_file_name);
            
            if(eePos_traj2d.size() < knot_points){std::cout << "precomputed traj length < knotpoints, not implemented\n"; continue; }


            std::vector<pcg_t> h_eePos_traj;
            for (const auto& vec : eePos_traj2d) {
                h_eePos_traj.insert(h_eePos_traj.end(), vec.begin(), vec.end());
            }
            std::vector<pcg_t> h_xu_traj;
            for (const auto& xu_vec : xu_traj2d) {
                h_xu_traj.insert(h_xu_traj.end(), xu_vec.begin(), xu_vec.end());
            }

            gpuErrchk(hipMalloc(&d_eePos_traj, h_eePos_traj.size()*sizeof(pcg_t)));
            gpuErrchk(hipMemcpy(d_eePos_traj, h_eePos_traj.data(), h_eePos_traj.size()*sizeof(pcg_t), hipMemcpyHostToDevice));
            
            gpuErrchk(hipMalloc(&d_xu_traj, h_xu_traj.size()*sizeof(pcg_t)));
            gpuErrchk(hipMemcpy(d_xu_traj, h_xu_traj.data(), h_xu_traj.size()*sizeof(pcg_t), hipMemcpyHostToDevice));
            
            gpuErrchk(hipMalloc(&d_xs, state_size*sizeof(pcg_t)));
            gpuErrchk(hipMemcpy(d_xs, h_xu_traj.data(), state_size*sizeof(pcg_t), hipMemcpyHostToDevice));


            track<pcg_t>(state_size, control_size, knot_points, static_cast<uint32_t>(eePos_traj2d.size()), timestep, d_eePos_traj, d_xu_traj, d_xs, start_state, goal_state, single_traj_test_iter);

            gpuErrchk(hipPeekAtLastError());
            
        }
        break;
    }



    gpuErrchk(hipFree(d_xu_traj));
    gpuErrchk(hipFree(d_eePos_traj));
    gpuErrchk(hipFree(d_xs));

    return 0;
}