#include <fstream>
#include <vector>
#include <sstream>
#include <iostream>
#include "toplevel.cuh"

#include "iiwa_plant.cuh"







/*

pre-compute trajectory

loop:
    solve from current state vector
    simulate for solve duration with old traj


-------------------------------------------

realtime

pre-compute trajectory

concurrently:
    track traj:
        shift traj with new state vector
        solve QP
    move:
        follow traj
        on traj update: 


*/





std::vector<std::vector<float>> readCSVToVecVec(const std::string& filename) {
    std::vector<std::vector<float>> data;
    std::ifstream infile(filename);

    if (!infile.is_open()) {
        std::cerr << "File could not be opened!\n";
    } else {
        std::string line;


        while (std::getline(infile, line)) {
            std::vector<float> row;
            std::stringstream ss(line);
            std::string val;

            while (std::getline(ss, val, ',')) {
                row.push_back(std::stof(val));
            }

            data.push_back(row);
        }
    }

    infile.close();
    return data;
}



int main(){
    const uint32_t state_size = 14;
    const uint32_t control_size = 7;
    const uint32_t knot_points = 50;


    std::vector<std::vector<float>> traj2d = readCSVToVecVec("./testfiles/0_traj.csv");
    std::vector<std::vector<float>> lambdas2d = readCSVToVecVec("./testfiles/0_lambdas.csv");

    std::vector<float> h_traj;
    for (const auto& vec : traj2d) {
        h_traj.insert(h_traj.end(), vec.begin(), vec.end());
    }
    std::vector<float> h_lambdas;
    for (const auto& vec : lambdas2d) {
        h_lambdas.insert(h_lambdas.end(), vec.begin(), vec.end());
    }

    // std::cout << "pre-computed traj\n";
    // for (int i = 0; i < traj2d.size(); i++){
    //     for (int j = 0; j < traj2d[i].size(); j++){
    //         std::cout << traj2d[i][j] << " ";
    //     }
    //     std::cout << std::endl;
    // }
    // std::cout << "pre-computed lambdas\n";
    // for (int i = 0; i < lambdas2d.size(); i++){
    //     for (int j = 0; j < lambdas2d[i].size(); j++){
    //         std::cout << lambdas2d[i][j] << " ";
    //     }
    //     std::cout << std::endl;
    // }

    float *d_traj, *d_traj_lambdas, *d_xs;
    gpuErrchk(hipMalloc(&d_traj, h_traj.size()*sizeof(float)));
    gpuErrchk(hipMemcpy(d_traj, h_traj.data(), h_traj.size()*sizeof(float), hipMemcpyHostToDevice));
    
    gpuErrchk(hipMalloc(&d_xs, state_size*sizeof(float)));
    gpuErrchk(hipMemcpy(d_xs, h_traj.data(), state_size*sizeof(float), hipMemcpyHostToDevice));
    
    gpuErrchk(hipMalloc(&d_traj_lambdas, h_lambdas.size() *sizeof(float)));
    gpuErrchk(hipMemcpy(d_traj_lambdas, h_lambdas.data(), h_lambdas.size()*sizeof(float), hipMemcpyHostToDevice));


    track<float>(state_size, control_size, knot_points, traj2d.size(), .01, d_traj, d_traj_lambdas, d_xs);
    gpuErrchk(hipPeekAtLastError());
   


    gpuErrchk(hipFree(d_traj));
    gpuErrchk(hipFree(d_xs));
    gpuErrchk(hipFree(d_traj_lambdas))

    return 0;
}