#include <iostream>
#include "toplevel.cuh"

#include "iiwa_plant.cuh"







/*

pre-compute trajectory

loop:
    shift trajectory with new state vector
    solve from current state vector
    simulate for solve duration


-------------------------------------------

realtime

pre-compute trajectory

concurrently:
    track traj:
        shift traj with new state vector
        solve QP
    move:
        follow traj
        on traj update: 

*/





int main(){
    const uint32_t state_size = 14;
    const uint32_t control_size = 7;
    const uint32_t knot_points = 3;

    float h_traj[56] = {-0.7072,    0.7312,   -1.4039,    0.2646,    1.4471,    1.575 ,    0.5363,    0.215 , 0.0392,   -4.8188,    2.9574,    1.3779,    0.7255,    1.0542, -112.371 ,  162.7095, -21.7513,  -35.1634,   -0.0134,    3.246 ,    0.1966, -0.2716,    0.4929,   -0.9196,    0.0426,    0.7934,    1.5672,    0.0077,   -1.5472, 0.3976,    0.3557,    1.5639,    0.5326,    0.9623,    1.0188, -112.371 ,  162.7095, -21.7513,  -35.1634,   -0.0134,    3.246 ,    0.1966,  -0.4264,   0.5326,  -0.884 ,   0.199 ,   0.8466,   1.6634,   0.1096, -28.0848,   9.7976, -42.5967,  33.0413,  57.6764,  23.3162,  18.1502};
    float h_xu[56] = {-0.63498, 0.76645, -1.3094, 0.29421, 1.52768, 1.61919, 0.54144, 0.30833, 0.08429, -4.7648, 3.02619, 1.45485, 0.72795, 1.09755, -112.27, 162.742, -21.678, -35.113, 0.00713, 3.26368, 0.19843, -0.2607, 0.58379, -0.8371, 0.10687, 0.86277, 1.64689, 0.07752, -1.5099, 0.42137, 0.42680, 1.59879, 0.57870, 1.05100, 1.03166, -112.33, 162.733, -21.728, -35.125, 0.03702, 3.30164, 0.28213, -0.3508, 0.59656, -0.8614, 0.25919, 0.92122, 1.70438, 0.18908, -28.054, 9.85011, -42.577, 33.0433, 57.7261, 23.4036, 18.1905};
    float h_lambda[42] = {-5.0946,    61.5739,  -191.5559,   361.1493,     5.8803,    -7.007 ,     0.7601,    -2.05  ,     2.5814,   -17.9229,    34.9526,     0.6137,    -0.6097,     0.0788,    -5.0341,    79.4689,  -172.678 ,   333.9097,     6.2139,    -5.4741,     0.7087,    -0.6474,    -5.1649,     0.3811,    -0.1262,    -0.0873,    -0.0163,    -0.    ,    -4.9748,   142.6363,    21.6536,   -46.603 ,     2.3691,     1.1642,     0.6466,    -1.1772,   -18.8751,    -0.1228,     3.4542,    -0.2771,    -0.0852,     0.0039};

    std::cout << "INITIAL XU\n";
    for(int i = 0; i < 3; i++){
        for(int j = 0; j < 21; j++){
            if(i==2 && j > 13){ continue; }
            std::cout << h_xu[21 * i + j] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << "INITIAL TRAJ\n";
    for(int i = 0; i < 3; i++){
        for(int j = 0; j < 21; j++){
            if(i==2 && j > 13){ continue; }
            std::cout << h_traj[21 * i + j] << " ";
        }
        std::cout << std::endl;
    }

    float *d_traj, *d_lambda, *d_xu;
    gpuErrchk(hipMalloc(&d_traj, 56 * sizeof(float)));
    gpuErrchk(hipMemcpy(d_traj, h_traj, 56*sizeof(float), hipMemcpyHostToDevice));
    
    gpuErrchk(hipMalloc(&d_xu, 56*sizeof(float)));
    gpuErrchk(hipMemcpy(d_xu, h_xu, 56*sizeof(float), hipMemcpyHostToDevice));
    
    gpuErrchk(hipMalloc(&d_lambda, 42*sizeof(float)));
    gpuErrchk(hipMemcpy(d_lambda, h_lambda, 42*sizeof(float), hipMemcpyHostToDevice));
    

    track<float>(state_size, control_size, knot_points, .1, d_xu, d_traj, d_lambda);
    gpuErrchk(hipPeekAtLastError());
   
    std::cout << "FINAL xu\n";
    float h_xu_copy[56];
    gpuErrchk(hipMemcpy(h_xu_copy, d_xu, 56*sizeof(float), hipMemcpyDeviceToHost));
    for(int i = 0; i < 3; i++){
        for(int j = 0; j < 21; j++){
            if(i==2 && j > 13){ continue; }
            std::cout << h_xu_copy[21 * i + j] << " ";
        }
        std::cout << std::endl;
    }


    gpuErrchk(hipFree(d_traj));
    gpuErrchk(hipFree(d_xu));
    gpuErrchk(hipFree(d_lambda));

    return 0;
}