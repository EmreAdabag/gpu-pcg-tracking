#include <fstream>
#include <vector>
#include <sstream>
#include <iostream>
#include "toplevel.cuh"
#include "qdldl.h"
#include "rbd_plant.cuh"
#include "settings.cuh"






template <typename T>
std::vector<std::vector<T>> readCSVToVecVec(const std::string& filename) {
    std::vector<std::vector<T>> data;
    std::ifstream infile(filename);

    if (!infile.is_open()) {
        std::cerr << "File [ " << filename << " ] could not be opened!\n";
    } else {
        std::string line;


        while (std::getline(infile, line)) {
            std::vector<T> row;
            std::stringstream ss(line);
            std::string val;

            while (std::getline(ss, val, ',')) {
                row.push_back(std::stof(val));
            }

            data.push_back(row);
        }
    }

    infile.close();
    return data;
}




int main(){

    const uint32_t state_size = grid::NUM_JOINTS*2;
    const uint32_t control_size = grid::NUM_JOINTS;
    const uint32_t knot_points = KNOT_POINTS;
    const pcg_t timestep = .015625;

    const uint32_t traj_test_iters = 1;

    // checks GPU space for pcg
    checkPcgOccupancy<pcg_t>((void *) pcg<pcg_t, state_size, knot_points>, PCG_NUM_THREADS, state_size, knot_points);    
    if(!std::is_same<QDLDL_float, pcg_t>::value){ std::cout << "GPU-PCG QDLDL type mismatch" << std::endl; exit(1); }

    print_test_config();

    const uint32_t recorded_states = 5;
    const uint32_t start_goal_combinations = recorded_states*recorded_states;

    char traj_file_name[100];
    // char lambda_file_name[100];

    int start_state, goal_state;
    pcg_t *d_traj, *d_xs;

    for(uint32_t ind = 0; ind < start_goal_combinations; ind++){

        start_state = ind % recorded_states;
        goal_state = ind / recorded_states;
        if(start_state == goal_state && start_state != 0){ continue; }
        std::cout << "start: " << start_state << " goal: " << goal_state << std::endl;

        for (int single_traj_test_iter = 0; single_traj_test_iter < traj_test_iters; single_traj_test_iter++){

            // read in traj
            snprintf(traj_file_name, sizeof(traj_file_name), "testfiles/%d_%d_traj.csv", start_state, goal_state);
            std::vector<std::vector<pcg_t>> traj2d = readCSVToVecVec<pcg_t>(traj_file_name);

            if(traj2d.size() < knot_points){std::cout << "precomputed traj length < knot points, not implemented\n"; continue; }

            std::vector<pcg_t> h_traj;
            for (const auto& vec : traj2d) {
                h_traj.insert(h_traj.end(), vec.begin(), vec.end());
            }

            gpuErrchk(hipMalloc(&d_traj, h_traj.size()*sizeof(pcg_t)));
            gpuErrchk(hipMemcpy(d_traj, h_traj.data(), h_traj.size()*sizeof(pcg_t), hipMemcpyHostToDevice));
            
            gpuErrchk(hipMalloc(&d_xs, state_size*sizeof(pcg_t)));
            gpuErrchk(hipMemcpy(d_xs, h_traj.data(), state_size*sizeof(pcg_t), hipMemcpyHostToDevice));
            

            track<pcg_t>(state_size, control_size, knot_points, static_cast<uint32_t>(traj2d.size()), timestep, d_traj, d_xs, start_state, goal_state, single_traj_test_iter);

            gpuErrchk(hipPeekAtLastError());
            
        }
        break;
    }



    gpuErrchk(hipFree(d_traj));
    gpuErrchk(hipFree(d_xs));

    return 0;
}